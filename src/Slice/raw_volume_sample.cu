#include "hip/hip_runtime.h"
#include "raw_volume_sample.cuh"
#include "Algorithm/hip/hip_vector_types.h"
#include "Common/cuda_utils.hpp"
VS_START



__constant__ RawSampleParameter sampleParameter;//use for kernel function


__global__ void CUDARawVolumeSample(uint8_t* image,//output result
                                    hipTextureObject_t  volume_data//cuda texture for volume data
                                    );


void CUDARawVolumeSampler::SetVolumeData(uint8_t *data, uint32_t dim_x, uint32_t dim_y, uint32_t dim_z) {
    CUDA_DRIVER_API_CALL(hipCtxSetCurrent(cu_ctx));
    assert(data && dim_x && dim_y && dim_z);
    this->volume_x=dim_x;
    this->volume_y=dim_y;
    this->volume_z=dim_z;
    this->volume_data_size=(size_t)dim_x*dim_y*dim_z;


    CreateCUDATexture3D(make_hipExtent(dim_x,dim_y,dim_z),&cu_volume_data,
                        &volume_texture);
    UpdateCUDATexture3D(data,cu_volume_data,dim_x,dim_y,dim_z,0,0,0);

    spdlog::info("Successfully set volume data to CUDA.");
}


void CUDARawVolumeSampler::Sample(uint8_t *data, Slice slice,float space_x,float space_y,float space_z) {
    //todo multi cuda context should call this function like opengl
    CUDA_DRIVER_API_CALL(hipCtxSetCurrent(cu_ctx));
    int w=slice.n_pixels_width;
    int h=slice.n_pixels_height;
    if(w!=old_w || h!=old_h){
        if(cu_sample_result)
            CUDA_RUNTIME_API_CALL(hipFree(cu_sample_result));
        CUDA_RUNTIME_API_CALL(hipMalloc((void**)&cu_sample_result,(size_t)w*h));
    }
    assert(cu_sample_result);
    old_w=w;
    old_h=h;

    RawSampleParameter sample_parameter;
    sample_parameter.image_w=w;
    sample_parameter.image_h=h;
    sample_parameter.origin=make_float3(slice.origin[0],slice.origin[1],slice.origin[2]);
    sample_parameter.right=make_float3(slice.right[0],slice.right[1],slice.right[2]);
    sample_parameter.down=make_float3(-slice.up[0],-slice.up[1],-slice.up[2]);
    sample_parameter.voxels_per_pixel=make_float2(slice.voxel_per_pixel_width,slice.voxel_per_pixel_height);
    sample_parameter.volume_board=make_float3(volume_x,volume_y,volume_z);
    sample_parameter.space=make_float3(space_x,space_y,space_z);
    sample_parameter.base_space=std::min({space_x,space_y,space_z});
    CUDA_RUNTIME_API_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sampleParameter),&sample_parameter,sizeof(RawSampleParameter)));

    dim3 threads_per_block={16,16};
    dim3 blocks_per_grid={(w+threads_per_block.x-1)/threads_per_block.x,(h+threads_per_block.y-1)/threads_per_block.y};

    CUDARawVolumeSample<<<blocks_per_grid,threads_per_block>>>(cu_sample_result,volume_texture);
    CUDA_RUNTIME_CHECK

    CUDA_RUNTIME_API_CALL(hipMemcpy(data,cu_sample_result,(size_t)w*h,hipMemcpyDefault));

    spdlog::info("Finish CUDA raw volume sample.");
}

__global__ void CUDARawVolumeSample(uint8_t *image, hipTextureObject_t volume_data) {
    int image_x=blockIdx.x*blockDim.x+threadIdx.x;
    int image_y=blockIdx.y*blockDim.y+threadIdx.y;
    if(image_x>=sampleParameter.image_w || image_y>=sampleParameter.image_h) return;
    uint64_t image_idx=(uint64_t)image_y*sampleParameter.image_w+image_x;

    float3 virtual_sample_pos=sampleParameter.origin+((image_x-(int)sampleParameter.image_w/2)*sampleParameter.voxels_per_pixel.x*sampleParameter.right
                                                    +(image_y-(int)sampleParameter.image_h/2)*sampleParameter.voxels_per_pixel.y*sampleParameter.down)
                                                     *sampleParameter.base_space/sampleParameter.space   ;



    float3 physical_sample_pos=virtual_sample_pos/ sampleParameter.volume_board;


    image[image_idx]=tex3D<float>(volume_data,physical_sample_pos.x,physical_sample_pos.y,physical_sample_pos.z)*255;
//    image[image_idx]=255;
}


VS_END
